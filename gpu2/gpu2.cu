#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "gpu2.cuh"
#include "../common/utils.hpp"

#define PAD_TOP 2
#define PAD_LEFT 2
#define PAD_BOTTOM 1
#define PAD_RIGHT 1

#define P2(d_data, row, col, width) ((d_data)[((row)-1) * (width) +  (col)   ])
#define P3(d_data, row, col, width) ((d_data)[((row)-1) * (width) + ((col)-1)])
#define P4(d_data, row, col, width) ((d_data)[ (row)    * (width) + ((col)-1)])
#define P5(d_data, row, col, width) ((d_data)[((row)+1) * (width) + ((col)-1)])
#define P6(d_data, row, col, width) ((d_data)[((row)+1) * (width) +  (col)   ])
#define P7(d_data, row, col, width) ((d_data)[((row)+1) * (width) + ((col)+1)])
#define P8(d_data, row, col, width) ((d_data)[ (row)    * (width) + ((col)+1)])
#define P9(d_data, row, col, width) ((d_data)[((row)-1) * (width) + ((col)+1)])

// Computes the number of black neighbors around a pixel.
__device__ uint8_t black_neighbors_around(uint8_t* d_data, unsigned int row, unsigned int col, unsigned int width) {
    uint8_t count = 0;

    count += (P2(d_data, row, col, width) == BINARY_BLACK);
    count += (P3(d_data, row, col, width) == BINARY_BLACK);
    count += (P4(d_data, row, col, width) == BINARY_BLACK);
    count += (P5(d_data, row, col, width) == BINARY_BLACK);
    count += (P6(d_data, row, col, width) == BINARY_BLACK);
    count += (P7(d_data, row, col, width) == BINARY_BLACK);
    count += (P8(d_data, row, col, width) == BINARY_BLACK);
    count += (P9(d_data, row, col, width) == BINARY_BLACK);

    return count;
}

// Performs an image skeletonization algorithm on the input Bitmap, and stores
// the result in the output Bitmap.
unsigned int skeletonize(Bitmap** src_bitmap, Bitmap** dst_bitmap, Padding padding, dim3 grid_dim, dim3 block_dim) {
    // allocate memory on device
    uint8_t* d_src_data = NULL;
    uint8_t* d_dst_data = NULL;
    unsigned int data_size = (*src_bitmap)->width * (*src_bitmap)->height * sizeof(uint8_t);
    hipError_t d_src_malloc_success = hipMalloc((void**) &d_src_data, data_size);
    hipError_t d_dst_malloc_success = hipMalloc((void**) &d_dst_data, data_size);
    assert((d_src_malloc_success == hipSuccess) && "Error: could not allocate memory for d_src_data");
    assert((d_dst_malloc_success == hipSuccess) && "Error: could not allocate memory for d_dst_data");

    // send data to device
    hipMemcpy(d_src_data, (*src_bitmap)->data, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dst_data, (*dst_bitmap)->data, data_size, hipMemcpyHostToDevice);

    unsigned int iterations = 0;
    do {
        skeletonize_pass<<<grid_dim, block_dim, (block_dim.x + padding.left + padding.right) * (block_dim.y + padding.top + padding.bottom) * sizeof(uint8_t)>>>(d_src_data, d_dst_data, (*src_bitmap)->width, padding);

        // bring data back from device
        hipMemcpy((*src_bitmap)->data, d_src_data, data_size, hipMemcpyDeviceToHost);
        hipMemcpy((*dst_bitmap)->data, d_dst_data, data_size, hipMemcpyDeviceToHost);

        swap_bitmaps((void**) &d_src_data, (void**) &d_dst_data);

        iterations++;
        printf(".");
        fflush(stdout);
    } while (!are_identical_bitmaps(*src_bitmap, *dst_bitmap));

    // free memory on device
    hipFree(d_src_data);
    hipFree(d_dst_data);

    return iterations;
}

// Performs 1 iteration of the thinning algorithm.
__global__ void skeletonize_pass(uint8_t* d_src, uint8_t* d_dst, unsigned int width, Padding padding) {
    // shared memory for d_src tile
    extern __shared__ uint8_t s_src[];

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    unsigned int bdx = blockDim.x;
    unsigned int bdy = blockDim.y;

    unsigned int row = by * bdy + ty + padding.top;
    unsigned int col = bx * bdx + tx + padding.left;

    // load a tile of d_src into s_src

    if (((tx % bdx) == 0) & ((ty % bdy) == 0)) {
        // top-left
    } else if (((tx % bdx) == (bdx - 1)) & ((ty % bdy) == 0)) {
        // top-right
    } else if (((tx % bdx) == 0) & ((ty % bdy) == (bdy - 1))) {
        // bottom-left
    } else if (((tx % bdx) == (bdx - 1)) & ((ty % bdy) == (bdy - 1))) {
        // bottom-right
    } else if ((ty % bdy) == 0) {
        // top-center
    } else if ((ty % bdy) == (bdy - 1)) {
        // bottom-center
    } else {
        // center-center
    }

    // make sure all threads have finished loading their data into shared memory
    __syncthreads();

    uint8_t NZ = black_neighbors_around(d_src, row, col, width);
    uint8_t TR_P1 = wb_transitions_around(d_src, row, col, width);
    uint8_t TR_P2 = wb_transitions_around(d_src, row-1, col, width);
    uint8_t TR_P4 = wb_transitions_around(d_src, row, col-1, width);
    uint8_t P2 = P2(d_src, row, col, width);
    uint8_t P4 = P4(d_src, row, col, width);
    uint8_t P6 = P6(d_src, row, col, width);
    uint8_t P8 = P8(d_src, row, col, width);

    uint8_t thinning_cond_1 = ((2 <= NZ) & (NZ <= 6));
    uint8_t thinning_cond_2 = (TR_P1 == 1);
    uint8_t thinning_cond_3 = (((P2 & P4 & P8) == 0) | (TR_P2 != 1));
    uint8_t thinning_cond_4 = (((P2 & P4 & P6) == 0) | (TR_P4 != 1));
    uint8_t thinning_cond_ok = thinning_cond_1 & thinning_cond_2 & thinning_cond_3 & thinning_cond_4;

    d_dst[row * width + col] = BINARY_WHITE + ((1 - thinning_cond_ok) * d_src[row * width + col]);
}

// Computes the number of white to black transitions around a pixel.
__device__ uint8_t wb_transitions_around(uint8_t* d_data, unsigned int row, unsigned int col, unsigned int width) {
    uint8_t count = 0;

    count += ( (P2(d_data, row, col, width) == BINARY_WHITE) & (P3(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P3(d_data, row, col, width) == BINARY_WHITE) & (P4(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P4(d_data, row, col, width) == BINARY_WHITE) & (P5(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P5(d_data, row, col, width) == BINARY_WHITE) & (P6(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P6(d_data, row, col, width) == BINARY_WHITE) & (P7(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P7(d_data, row, col, width) == BINARY_WHITE) & (P8(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P8(d_data, row, col, width) == BINARY_WHITE) & (P9(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P9(d_data, row, col, width) == BINARY_WHITE) & (P2(d_data, row, col, width) == BINARY_BLACK) );

    return count;
}

int main(int argc, char** argv) {
    assert(argc == 5 && "Usage: gpu1 <input_file_name.bmp> <output_file_name.bmp> <block_dim_x> <block_dim_y>");

    char* src_fname = argv[1];
    char* dst_fname = argv[2];
    char* block_dim_x_string = argv[3];
    char* block_dim_y_string = argv[4];

    printf("src_fname   = %s\n", src_fname);
    printf("dst_fname   = %s\n", dst_fname);
    printf("block dim X = %s\n", block_dim_x_string);
    printf("block dim Y = %s\n", block_dim_y_string);

    // load src image
    Bitmap* src_bitmap = loadBitmap(src_fname);
    assert(src_bitmap != NULL && "Error: could not load src bitmap");

    // validate src image is 8-bit binary-valued grayscale image
    assert(is_binary_valued_grayscale_image(src_bitmap) && "Error: Only 8-bit binary-valued grayscale images are supported. Values must be black (0) or white (255) only");

    // we work on true binary images
    grayscale_to_binary(src_bitmap);

    // Create dst bitmap image (empty for now)
    Bitmap* dst_bitmap = createBitmap(src_bitmap->width, src_bitmap->height, src_bitmap->depth);
    assert(dst_bitmap != NULL && "Error: could not allocate memory for dst bitmap");

    // Dimensions of computing elements on the CUDA device.
    // Computing the grid dimensions depends on PAD_TOP and PAD_LEFT.
    unsigned int block_dim_x = strtol(block_dim_x_string, NULL, 10);
    unsigned int block_dim_y = strtol(block_dim_y_string, NULL, 10);
    unsigned int grid_dim_x = (unsigned int) ceil((src_bitmap->width) / ((double) block_dim_x));
    unsigned int grid_dim_y = (unsigned int) ceil((src_bitmap->height)/ ((double) block_dim_y));
    dim3 block_dim(block_dim_x, block_dim_y);
    dim3 grid_dim(grid_dim_x, grid_dim_y);

    // Pad the binary images with pixels on each side. This will be useful when
    // implementing the skeletonization algorithm, because the mask we use
    // depends on P2 and P4, which also have their own window.
    // ATTENTION : it is important to use cast to (int) since we want to test
    // for a maximum value and the subtraction can yield a negative number.
    Padding padding;
    padding.top = PAD_TOP;
    padding.bottom = max((int) ((grid_dim_y * block_dim_y) - (src_bitmap->height + PAD_BOTTOM)), PAD_BOTTOM);
    padding.left = PAD_LEFT;
    padding.right = max((int) ((grid_dim_x * block_dim_x) - (src_bitmap->width + PAD_RIGHT)), PAD_RIGHT);
    pad_binary_bitmap(&src_bitmap, BINARY_WHITE, padding);
    pad_binary_bitmap(&dst_bitmap, BINARY_WHITE, padding);

    unsigned int iterations = skeletonize(&src_bitmap, &dst_bitmap, padding, grid_dim, block_dim);
    printf(" %u iterations\n", iterations);

    // Remove extra padding that was added to the images (don't care about
    // src_bitmap, so only need to unpad dst_bitmap)
    unpad_binary_bitmap(&dst_bitmap, padding);

    // save 8-bit binary-valued grayscale version of dst_bitmap to dst_fname
    binary_to_grayscale(dst_bitmap);
    int save_successful = saveBitmap(dst_fname, dst_bitmap);
    assert(save_successful == 1 && "Error: could not save dst bitmap");

    // deallocate memory used for bitmaps
    free(src_bitmap);
    free(dst_bitmap);

    return EXIT_SUCCESS;
}
