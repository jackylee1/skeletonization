#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "gpu2.cuh"
#include "../common/gpu_only_utils.cuh"
#include "../common/utils.hpp"

void and_reduction(uint8_t* d_data, int width, int height, dim3 grid_dim, dim3 block_dim) {
    int shared_mem_size = block_dim.x * block_dim.y * sizeof(uint8_t);

    // iterative reductions of d_data
    do {
        and_reduction<<<grid_dim, block_dim, shared_mem_size>>>(d_data, width, height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        width = grid_dim.x;
        height = grid_dim.y;
        grid_dim.x = ceil(grid_dim.x / ((double) block_dim.x));
        grid_dim.y = ceil(grid_dim.y / ((double) block_dim.y));
    } while ((width * height) != 1);
}

// Adapted for 2D arrays from Nvidia cuda SDK samples
__global__ void and_reduction(uint8_t* d_data, int width, int height) {
    // shared memory for tile
    extern __shared__ uint8_t s_data[];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Load equality values into shared memory tile. We use 1 as the default
    // value since it is a binary AND reduction
    s_data[tid] = ((row < height) & (col < width)) ? d_data[row * width + col] : 1;
    __syncthreads();

    // do reduction in shared memory
    for (int s = ((blockDim.x * blockDim.y) / 2); s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] &= s_data[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        d_data[blockIdx.y * gridDim.x + blockIdx.x] = s_data[0];
    }
}

// Computes the number of black neighbors around a pixel.
__device__ uint8_t black_neighbors_around(uint8_t* d_data, int row, int col, int width, int height) {
    uint8_t count = 0;

    count += (P2_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P3_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P4_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P5_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P6_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P7_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P8_f(d_data, row, col, width, height) == BINARY_BLACK);
    count += (P9_f(d_data, row, col, width, height) == BINARY_BLACK);

    return count;
}

__device__ uint8_t is_outside_image(int row, int col, int width, int height) {
    return (row < 0) | (row > (height - 1)) | (col < 0) | (col > (width - 1));
}

__device__ uint8_t P2_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row - 1, col, width, height) ? BINARY_WHITE : data[(row - 1) * width + col];
}

__device__ uint8_t P3_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row - 1, col - 1, width, height) ? BINARY_WHITE : data[(row - 1) * width + (col - 1)];
}

__device__ uint8_t P4_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row, col - 1, width, height) ? BINARY_WHITE : data[row * width + (col - 1)];
}

__device__ uint8_t P5_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row + 1, col - 1, width, height) ? BINARY_WHITE : data[(row + 1) * width + (col - 1)];
}

__device__ uint8_t P6_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row + 1, col, width, height) ? BINARY_WHITE : data[(row + 1) * width + col];
}

__device__ uint8_t P7_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row + 1, col + 1, width, height) ? BINARY_WHITE : data[(row + 1) * width + (col + 1)];
}

__device__ uint8_t P8_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row, col + 1, width, height) ? BINARY_WHITE : data[row * width + (col + 1)];
}

__device__ uint8_t P9_f(uint8_t* data, int row, int col, int width, int height) {
    return is_outside_image(row - 1, col + 1, width, height) ? BINARY_WHITE : data[(row - 1) * width + (col + 1)];
}

__global__ void pixel_equality(uint8_t* d_in_1, uint8_t* d_in_2, uint8_t* d_out, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height) & (col < width)) {
        d_out[row * width + col] = (d_in_1[row * width + col] == d_in_2[row * width + col]);
    }
}

// Performs an image skeletonization algorithm on the input Bitmap, and stores
// the result in the output Bitmap.
int skeletonize(Bitmap** src_bitmap, Bitmap** dst_bitmap, dim3 grid_dim, dim3 block_dim) {
    // allocate memory on device
    uint8_t* d_src_data = NULL;
    uint8_t* d_dst_data = NULL;
    uint8_t* d_equ_data = NULL;
    int data_size = (*src_bitmap)->width * (*src_bitmap)->height * sizeof(uint8_t);
    gpuErrchk(hipMalloc((void**) &d_src_data, data_size));
    gpuErrchk(hipMalloc((void**) &d_dst_data, data_size));
    gpuErrchk(hipMalloc((void**) &d_equ_data, data_size));

    // send data to device
    gpuErrchk(hipMemcpy(d_src_data, (*src_bitmap)->data, data_size, hipMemcpyHostToDevice));

    uint8_t are_identical_bitmaps = 0;
    int iterations = 0;
    do {
        skeletonize_pass<<<grid_dim, block_dim>>>(d_src_data, d_dst_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        pixel_equality<<<grid_dim, block_dim>>>(d_src_data, d_dst_data, d_equ_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        and_reduction(d_equ_data, (*src_bitmap)->width, (*src_bitmap)->height, grid_dim, block_dim);

        // bring reduced bitmap equality information back from device
        gpuErrchk(hipMemcpy(&are_identical_bitmaps, d_equ_data, 1 * sizeof(uint8_t), hipMemcpyDeviceToHost));

        swap_bitmaps((void**) &d_src_data, (void**) &d_dst_data);

        iterations++;
        printf(".");
        fflush(stdout);
    } while (!are_identical_bitmaps);

    // bring dst_bitmap back from device
    gpuErrchk(hipMemcpy((*dst_bitmap)->data, d_dst_data, data_size, hipMemcpyDeviceToHost));

    // free memory on device
    gpuErrchk(hipFree(d_src_data));
    gpuErrchk(hipFree(d_dst_data));
    gpuErrchk(hipFree(d_equ_data));

    return iterations;
}

// Performs 1 iteration of the thinning algorithm.
__global__ void skeletonize_pass(uint8_t* d_src, uint8_t* d_dst, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height) & (col < width)) {
        uint8_t NZ = black_neighbors_around(d_src, row, col, width, height);
        uint8_t TR_P1 = wb_transitions_around(d_src, row, col, width, height);
        uint8_t TR_P2 = wb_transitions_around(d_src, row - 1, col, width, height);
        uint8_t TR_P4 = wb_transitions_around(d_src, row, col - 1, width, height);
        uint8_t P2 = P2_f(d_src, row, col, width, height);
        uint8_t P4 = P4_f(d_src, row, col, width, height);
        uint8_t P6 = P6_f(d_src, row, col, width, height);
        uint8_t P8 = P8_f(d_src, row, col, width, height);

        uint8_t thinning_cond_1 = ((2 <= NZ) & (NZ <= 6));
        uint8_t thinning_cond_2 = (TR_P1 == 1);
        uint8_t thinning_cond_3 = (((P2 & P4 & P8) == 0) | (TR_P2 != 1));
        uint8_t thinning_cond_4 = (((P2 & P4 & P6) == 0) | (TR_P4 != 1));
        uint8_t thinning_cond_ok = thinning_cond_1 & thinning_cond_2 & thinning_cond_3 & thinning_cond_4;

        d_dst[row * width + col] = BINARY_WHITE + ((1 - thinning_cond_ok) * d_src[row * width + col]);
    }
}

// Computes the number of white to black transitions around a pixel.
__device__ uint8_t wb_transitions_around(uint8_t* d_data, int row, int col, int width, int height) {
    uint8_t count = 0;

    count += ((P2_f(d_data, row, col, width, height) == BINARY_WHITE) & (P3_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P3_f(d_data, row, col, width, height) == BINARY_WHITE) & (P4_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P4_f(d_data, row, col, width, height) == BINARY_WHITE) & (P5_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P5_f(d_data, row, col, width, height) == BINARY_WHITE) & (P6_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P6_f(d_data, row, col, width, height) == BINARY_WHITE) & (P7_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P7_f(d_data, row, col, width, height) == BINARY_WHITE) & (P8_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P8_f(d_data, row, col, width, height) == BINARY_WHITE) & (P9_f(d_data, row, col, width, height) == BINARY_BLACK));
    count += ((P9_f(d_data, row, col, width, height) == BINARY_WHITE) & (P2_f(d_data, row, col, width, height) == BINARY_BLACK));

    return count;
}

int main(int argc, char** argv) {
    Bitmap* src_bitmap = NULL;
    Bitmap* dst_bitmap = NULL;
    dim3 grid_dim;
    dim3 block_dim;

    gpu_pre_skeletonization(argc, argv, &src_bitmap, &dst_bitmap, &grid_dim, &block_dim);

    int iterations = skeletonize(&src_bitmap, &dst_bitmap, grid_dim, block_dim);
    printf(" %u iterations\n", iterations);

    gpu_post_skeletonization(argv, &src_bitmap, &dst_bitmap);

    return EXIT_SUCCESS;
}
