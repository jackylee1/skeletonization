#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "gpu2.cuh"
#include "../common/gpu_only_utils.cuh"
#include "../common/utils.hpp"

#define P2(d_data, row, col, width) ((d_data)[((row) - 1) * (width) +  (col)     ])
#define P3(d_data, row, col, width) ((d_data)[((row) - 1) * (width) + ((col) - 1)])
#define P4(d_data, row, col, width) ((d_data)[ (row)      * (width) + ((col) - 1)])
#define P5(d_data, row, col, width) ((d_data)[((row) + 1) * (width) + ((col) - 1)])
#define P6(d_data, row, col, width) ((d_data)[((row) + 1) * (width) +  (col)     ])
#define P7(d_data, row, col, width) ((d_data)[((row) + 1) * (width) + ((col) + 1)])
#define P8(d_data, row, col, width) ((d_data)[ (row)      * (width) + ((col) + 1)])
#define P9(d_data, row, col, width) ((d_data)[((row) - 1) * (width) + ((col) + 1)])

void and_reduction(dim3 grid_dim, dim3 block_dim, uint8_t* d_pixel_equ, unsigned int pixel_equ_size) {
    unsigned int shared_mem_size = block_dim.x * block_dim.y * sizeof(uint8_t);
    unsigned int grid_size = grid_dim.x * grid_dim.y;
    unsigned int block_size = block_dim.x * block_dim.y;

    // iterative reductions of d_pixel_equ
    do {
        and_reduction<<<grid_size, block_size, shared_mem_size>>>(d_pixel_equ, pixel_equ_size);
        pixel_equ_size = grid_size;
        grid_size = ceil(grid_size / ((double) block_size));
    } while (pixel_equ_size != 1);
}

// Adapted from Nvidia cuda SDK samples
__global__ void and_reduction(uint8_t* d_data, unsigned int size) {
    // shared memory for tile (without padding, unlike in skeletonize_pass)
    extern __shared__ uint8_t s_data[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // load equality values into shared memory tile
    s_data[tid] = (i < size) ? d_data[i] : 1; // we use 1 since it is a binary AND
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = (blockDim.x / 2); s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] &= s_data[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        d_data[blockIdx.x] = s_data[0];
    }
}

// Computes the number of black neighbors around a pixel.
__device__ uint8_t black_neighbors_around(uint8_t* d_data, unsigned int row, unsigned int col, unsigned int width) {
    uint8_t count = 0;

    count += (P2(d_data, row, col, width) == BINARY_BLACK);
    count += (P3(d_data, row, col, width) == BINARY_BLACK);
    count += (P4(d_data, row, col, width) == BINARY_BLACK);
    count += (P5(d_data, row, col, width) == BINARY_BLACK);
    count += (P6(d_data, row, col, width) == BINARY_BLACK);
    count += (P7(d_data, row, col, width) == BINARY_BLACK);
    count += (P8(d_data, row, col, width) == BINARY_BLACK);
    count += (P9(d_data, row, col, width) == BINARY_BLACK);

    return count;
}

__global__ void pixel_equality(uint8_t* d_in_1, uint8_t* d_in_2, uint8_t* d_out, unsigned int width, Padding padding) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y + padding.top;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x + padding.left;

    d_out[(row - padding.top) * (width - padding.left - padding.right) + (col - padding.left)] = (d_in_1[row * width + col] == d_in_2[row * width + col]);
}

// Performs an image skeletonization algorithm on the input Bitmap, and stores
// the result in the output Bitmap.
unsigned int skeletonize(Bitmap** src_bitmap, Bitmap** dst_bitmap, Padding padding, dim3 grid_dim, dim3 block_dim) {
    // 1 byte of data indicating if the images are identical
    uint8_t grid_equ = 0;

    // allocate memory on device
    uint8_t* d_src_data = NULL;
    uint8_t* d_dst_data = NULL;
    uint8_t* d_pixel_equ = NULL;
    uint8_t* d_block_equ = NULL;
    uint8_t* d_grid_equ = NULL;

    unsigned int data_size = (*src_bitmap)->width * (*src_bitmap)->height * sizeof(uint8_t);
    unsigned int pixel_equ_size = ((*src_bitmap)->width - padding.left - padding.right) * ((*src_bitmap)->height - padding.top - padding.bottom) * sizeof(uint8_t);
    unsigned int block_equ_size = grid_dim.x * grid_dim.y * sizeof(uint8_t);
    unsigned int grid_equ_size = 1 * sizeof(uint8_t);

    hipError_t d_src_malloc_success = hipMalloc((void**) &d_src_data, data_size);
    hipError_t d_dst_malloc_success = hipMalloc((void**) &d_dst_data, data_size);
    hipError_t d_pixel_equ_malloc_success = hipMalloc((void**) &d_pixel_equ, pixel_equ_size);
    hipError_t d_block_equ_malloc_success = hipMalloc((void**) &d_block_equ, block_equ_size);
    hipError_t d_grid_equ_malloc_success = hipMalloc((void**) &d_grid_equ, grid_equ_size);

    assert((d_src_malloc_success == hipSuccess) && "Error: could not allocate memory for d_src_data");
    assert((d_dst_malloc_success == hipSuccess) && "Error: could not allocate memory for d_dst_data");
    assert((d_pixel_equ_malloc_success == hipSuccess) && "Error: could not allocate memory for d_pixel_equ");
    assert((d_block_equ_malloc_success == hipSuccess) && "Error: could not allocate memory for d_block_equ");
    assert((d_grid_equ_malloc_success == hipSuccess) && "Error: could not allocate memory for d_grid_equ");

    // send data to device
    hipMemcpy(d_src_data, (*src_bitmap)->data, data_size, hipMemcpyHostToDevice);

    // for dst_data, we don't need to actually send the real data. All we need
    // is to send some data that is correctly padded with BINARY_WHITE on the
    // sides.
    hipMemset(d_dst_data, BINARY_WHITE, data_size);

    unsigned int iterations = 0;
    do {
        // 2D grid & 2D block
        skeletonize_pass<<<grid_dim, block_dim>>>(d_src_data, d_dst_data, (*src_bitmap)->width, padding);
        pixel_equality<<<grid_dim, block_dim>>>(d_src_data, d_dst_data, d_pixel_equ, (*src_bitmap)->width, padding);

        // 1D grid & 1D block reduction
        and_reduction(grid_dim, block_dim, d_pixel_equ, pixel_equ_size);

        // bring d_grid_equ back from device
        hipMemcpy(&grid_equ, d_pixel_equ, grid_equ_size, hipMemcpyDeviceToHost);

        swap_bitmaps((void**) &d_src_data, (void**) &d_dst_data);

        iterations++;
        printf(".");
        fflush(stdout);
    } while (!grid_equ);

    // bring data back from device
    hipMemcpy((*dst_bitmap)->data, d_dst_data, data_size, hipMemcpyDeviceToHost);

    // free memory on device
    hipFree(d_src_data);
    hipFree(d_dst_data);
    hipFree(d_pixel_equ);
    hipFree(d_block_equ);
    hipFree(d_grid_equ);

    return iterations;
}

// Performs 1 iteration of the thinning algorithm.
__global__ void skeletonize_pass(uint8_t* d_src, uint8_t* d_dst, unsigned int width, Padding padding) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y + padding.top;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x + padding.left;

    uint8_t NZ = black_neighbors_around(d_src, row, col, width);
    uint8_t TR_P1 = wb_transitions_around(d_src, row, col, width);
    uint8_t TR_P2 = wb_transitions_around(d_src, row - 1, col, width);
    uint8_t TR_P4 = wb_transitions_around(d_src, row, col - 1, width);
    uint8_t P2 = P2(d_src, row, col, width);
    uint8_t P4 = P4(d_src, row, col, width);
    uint8_t P6 = P6(d_src, row, col, width);
    uint8_t P8 = P8(d_src, row, col, width);

    uint8_t thinning_cond_1 = ((2 <= NZ) & (NZ <= 6));
    uint8_t thinning_cond_2 = (TR_P1 == 1);
    uint8_t thinning_cond_3 = (((P2 & P4 & P8) == 0) | (TR_P2 != 1));
    uint8_t thinning_cond_4 = (((P2 & P4 & P6) == 0) | (TR_P4 != 1));
    uint8_t thinning_cond_ok = thinning_cond_1 & thinning_cond_2 & thinning_cond_3 & thinning_cond_4;

    d_dst[row * width + col] = BINARY_WHITE + ((1 - thinning_cond_ok) * d_src[row * width + col]);
}

// Computes the number of white to black transitions around a pixel.
__device__ uint8_t wb_transitions_around(uint8_t* d_data, unsigned int row, unsigned int col, unsigned int width) {
    uint8_t count = 0;

    count += ( (P2(d_data, row, col, width) == BINARY_WHITE) & (P3(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P3(d_data, row, col, width) == BINARY_WHITE) & (P4(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P4(d_data, row, col, width) == BINARY_WHITE) & (P5(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P5(d_data, row, col, width) == BINARY_WHITE) & (P6(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P6(d_data, row, col, width) == BINARY_WHITE) & (P7(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P7(d_data, row, col, width) == BINARY_WHITE) & (P8(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P8(d_data, row, col, width) == BINARY_WHITE) & (P9(d_data, row, col, width) == BINARY_BLACK) );
    count += ( (P9(d_data, row, col, width) == BINARY_WHITE) & (P2(d_data, row, col, width) == BINARY_BLACK) );

    return count;
}

int main(int argc, char** argv) {
    Bitmap* src_bitmap = NULL;
    Bitmap* dst_bitmap = NULL;
    Padding padding;
    dim3 grid_dim;
    dim3 block_dim;

    gpu_pre_skeletonization(argc, argv, &src_bitmap, &dst_bitmap, &padding, &grid_dim, &block_dim);

    unsigned int iterations = skeletonize(&src_bitmap, &dst_bitmap, padding, grid_dim, block_dim);
    printf(" %u iterations\n", iterations);

    gpu_post_skeletonization(argv, &src_bitmap, &dst_bitmap, &padding);

    return EXIT_SUCCESS;
}
