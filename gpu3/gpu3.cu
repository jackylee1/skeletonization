#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "gpu3.cuh"
#include "../common/gpu_only_utils.cuh"
#include "../common/lspbmp.hpp"
#include "../common/utils.hpp"

#define PAD_TOP 2
#define PAD_LEFT 2
#define PAD_BOTTOM 1
#define PAD_RIGHT 1

void and_reduction(uint8_t* g_data, int g_width, int g_height, dim3 grid_dim, dim3 block_dim) {
    int shared_mem_size = block_dim.x * block_dim.y * sizeof(uint8_t);

    // iterative reductions of g_data
    do {
        and_reduction<<<grid_dim, block_dim, shared_mem_size>>>(g_data, g_width, g_height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        g_width = grid_dim.x;
        g_height = grid_dim.y;
        grid_dim.x = ceil(grid_dim.x / ((double) block_dim.x));
        grid_dim.y = ceil(grid_dim.y / ((double) block_dim.y));
    } while ((g_width * g_height) != 1);
}

// Adapted for 2D arrays from Nvidia cuda SDK samples
__global__ void and_reduction(uint8_t* g_data, int g_width, int g_height) {
    // shared memory for tile
    extern __shared__ uint8_t s_data[];

    int g_row = blockIdx.y * blockDim.y + threadIdx.y;
    int g_col = blockIdx.x * blockDim.x + threadIdx.x;

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Load equality values into shared memory tile. We use 1 as the default
    // value, as it is an AND reduction
    s_data[tid] = is_outside_image(g_row, g_col, g_width, g_height) ? 1 : global_mem_read(g_data, g_row, g_col, g_width, g_height);;
    __syncthreads();

    // do reduction in shared memory
    for (int s = ((blockDim.x * blockDim.y) / 2); s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] &= s_data[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        uint8_t write_data = s_data[0];
        global_mem_write(g_data, blockIdx.y, blockIdx.x, gridDim.x, gridDim.y, write_data);
    }
}

// Computes the number of black neighbors around a pixel.
__device__ uint8_t black_neighbors_around(uint8_t* s_data, int s_row, int s_col, int s_width) {
    uint8_t count = 0;

    count += (P2_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P3_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P4_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P5_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P6_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P7_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P8_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);
    count += (P9_f(s_data, s_row, s_col, s_width) == BINARY_BLACK);

    return count;
}

__device__ uint8_t global_mem_read(uint8_t* g_data, int g_row, int g_col, int g_width, int g_height) {
    return is_outside_image(g_row, g_col, g_width, g_height) ? BINARY_WHITE : g_data[g_row * g_width + g_col];
}

__device__ void global_mem_write(uint8_t* g_data, int g_row, int g_col, int g_width, int g_height, uint8_t write_data) {
    if (!is_outside_image(g_row, g_col, g_width, g_height)) {
        g_data[g_row * g_width + g_col] = write_data;
    }
}

__device__ uint8_t is_outside_image(int g_row, int g_col, int g_width, int g_height) {
    return (g_row < 0) | (g_row > (g_height - 1)) | (g_col < 0) | (g_col > (g_width - 1));
}

__device__ void load_s_src(uint8_t* g_src, int g_row, int g_col, int g_width, int g_height, uint8_t* s_src, int s_row, int s_col, int s_width) {
    if ((threadIdx.y == 0) & (threadIdx.x == 0)) {
        // top-left corner
        s_src[(s_row - 2) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row - 2, g_col - 2, g_width, g_height);
        s_src[(s_row - 2) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row - 2, g_col - 1, g_width, g_height);
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(g_src, g_row - 2, g_col, g_width, g_height);

        s_src[(s_row - 1) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row - 1, g_col - 2, g_width, g_height);
        s_src[(s_row - 1) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row - 1, g_col - 1, g_width, g_height);
        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(g_src, g_row - 1, g_col, g_width, g_height);

        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row, g_col - 2, g_width, g_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row, g_col - 1, g_width, g_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
    } else if ((threadIdx.y == (blockDim.y - 1)) & (threadIdx.x == 0)) {
        // bottom-left corner
        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row, g_col - 2, g_width, g_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row, g_col - 1, g_width, g_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);

        s_src[(s_row + 1) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row + 1, g_col - 2, g_width, g_height);
        s_src[(s_row + 1) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row + 1, g_col - 1, g_width, g_height);
        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(g_src, g_row + 1, g_col, g_width, g_height);
    } else if ((threadIdx.y == (blockDim.y - 1)) & (threadIdx.x == (blockDim.x - 1))) {
        // bottom-right corner
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row, g_col + 1, g_width, g_height);

        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(g_src, g_row + 1, g_col, g_width, g_height);
        s_src[(s_row + 1) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row + 1, g_col + 1, g_width, g_height);
    } else if ((threadIdx.y == 0) & (threadIdx.x == (blockDim.x - 1))) {
        // top-right corner
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(g_src, g_row - 2, g_col, g_width, g_height);
        s_src[(s_row - 2) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row - 2, g_col + 1, g_width, g_height);

        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(g_src, g_row - 1, g_col, g_width, g_height);
        s_src[(s_row - 1) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row - 1, g_col + 1, g_width, g_height);

        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row, g_col + 1, g_width, g_height);
    } else if (threadIdx.y == 0) {
        // PAD_TOP top rows
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(g_src, g_row - 2, g_col, g_width, g_height);
        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(g_src, g_row - 1, g_col, g_width, g_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
    } else if (threadIdx.x == 0) {
        // PAD_LEFT left rows
        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(g_src, g_row, g_col - 2, g_width, g_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(g_src, g_row, g_col - 1, g_width, g_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
    } else if (threadIdx.y == (blockDim.y - 1)) {
        // PAD_BOTTOM bottom rows
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(g_src, g_row + 1, g_col, g_width, g_height);
    } else if (threadIdx.x == (blockDim.x - 1)) {
        // PAD_RIGHT right rows
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(g_src, g_row, g_col + 1, g_width, g_height);
    } else {
        // center pixels
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(g_src, g_row, g_col, g_width, g_height);
    }

    __syncthreads();
}

__device__ uint8_t P2_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row - 1) * s_width + s_col];
}

__device__ uint8_t P3_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row - 1) * s_width + (s_col - 1)];
}

__device__ uint8_t P4_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[s_row * s_width + (s_col - 1)];
}

__device__ uint8_t P5_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row + 1) * s_width + (s_col - 1)];
}

__device__ uint8_t P6_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row + 1) * s_width + s_col];
}

__device__ uint8_t P7_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row + 1) * s_width + (s_col + 1)];
}

__device__ uint8_t P8_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[s_row * s_width + (s_col + 1)];
}

__device__ uint8_t P9_f(uint8_t* s_data, int s_row, int s_col, int s_width) {
    return s_data[(s_row - 1) * s_width + (s_col + 1)];
}

__global__ void pixel_equality(uint8_t* g_in_1, uint8_t* g_in_2, uint8_t* g_out, int g_width, int g_height) {
    int g_row = blockIdx.y * blockDim.y + threadIdx.y;
    int g_col = blockIdx.x * blockDim.x + threadIdx.x;

    uint8_t write_data = (global_mem_read(g_in_1, g_row, g_col, g_width, g_height) == global_mem_read(g_in_2, g_row, g_col, g_width, g_height));
    global_mem_write(g_out, g_row, g_col, g_width, g_height, write_data);
}

// Performs an image skeletonization algorithm on the input Bitmap, and stores
// the result in the output Bitmap.
int skeletonize(Bitmap** src_bitmap, Bitmap** dst_bitmap, dim3 grid_dim, dim3 block_dim) {
    // allocate memory on device
    uint8_t* g_src_data = NULL;
    uint8_t* g_dst_data = NULL;
    uint8_t* g_equ_data = NULL;
    int g_data_size = (*src_bitmap)->width * (*src_bitmap)->height * sizeof(uint8_t);
    gpuErrchk(hipMalloc((void**) &g_src_data, g_data_size));
    gpuErrchk(hipMalloc((void**) &g_dst_data, g_data_size));
    gpuErrchk(hipMalloc((void**) &g_equ_data, g_data_size));

    // send data to device
    gpuErrchk(hipMemcpy(g_src_data, (*src_bitmap)->data, g_data_size, hipMemcpyHostToDevice));

    uint8_t are_identical_bitmaps = 0;
    int iterations = 0;
    do {
        int skeletonize_pass_shared_mem_size = (block_dim.x + PAD_LEFT + PAD_RIGHT) * (block_dim.y + PAD_TOP + PAD_BOTTOM) * sizeof(uint8_t);
        skeletonize_pass<<<grid_dim, block_dim, skeletonize_pass_shared_mem_size>>>(g_src_data, g_dst_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        pixel_equality<<<grid_dim, block_dim>>>(g_src_data, g_dst_data, g_equ_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        and_reduction(g_equ_data, (*src_bitmap)->width, (*src_bitmap)->height, grid_dim, block_dim);

        // bring reduced bitmap equality information back from device
        gpuErrchk(hipMemcpy(&are_identical_bitmaps, g_equ_data, 1 * sizeof(uint8_t), hipMemcpyDeviceToHost));

        swap_bitmaps((void**) &g_src_data, (void**) &g_dst_data);

        iterations++;
        printf(".");
        fflush(stdout);
    } while (!are_identical_bitmaps);

    // bring dst_bitmap back from device
    gpuErrchk(hipMemcpy((*dst_bitmap)->data, g_dst_data, g_data_size, hipMemcpyDeviceToHost));

    // free memory on device
    gpuErrchk(hipFree(g_src_data));
    gpuErrchk(hipFree(g_dst_data));
    gpuErrchk(hipFree(g_equ_data));

    return iterations;
}

// Performs 1 iteration of the thinning algorithm.
__global__ void skeletonize_pass(uint8_t* g_src, uint8_t* g_dst, int g_width, int g_height) {
    // shared memory for tile
    extern __shared__ uint8_t s_src[];

    int g_row = blockIdx.y * blockDim.y + threadIdx.y;
    int g_col = blockIdx.x * blockDim.x + threadIdx.x;

    int s_row = threadIdx.y + PAD_TOP;
    int s_col = threadIdx.x + PAD_LEFT;
    int s_width = blockDim.x + PAD_LEFT + PAD_RIGHT;

    // load data into shared memory
    load_s_src(g_src, g_row, g_col, g_width, g_height, s_src, s_row, s_col, s_width);

    uint8_t NZ = black_neighbors_around(s_src, s_row, s_col, s_width);
    uint8_t TR_P1 = wb_transitions_around(s_src, s_row, s_col, s_width);
    uint8_t TR_P2 = wb_transitions_around(s_src, s_row - 1, s_col, s_width);
    uint8_t TR_P4 = wb_transitions_around(s_src, s_row, s_col - 1, s_width);
    uint8_t P2 = P2_f(s_src, s_row, s_col, s_width);
    uint8_t P4 = P4_f(s_src, s_row, s_col, s_width);
    uint8_t P6 = P6_f(s_src, s_row, s_col, s_width);
    uint8_t P8 = P8_f(s_src, s_row, s_col, s_width);

    uint8_t thinning_cond_1 = ((2 <= NZ) & (NZ <= 6));
    uint8_t thinning_cond_2 = (TR_P1 == 1);
    uint8_t thinning_cond_3 = (((P2 & P4 & P8) == 0) | (TR_P2 != 1));
    uint8_t thinning_cond_4 = (((P2 & P4 & P6) == 0) | (TR_P4 != 1));
    uint8_t thinning_cond_ok = thinning_cond_1 & thinning_cond_2 & thinning_cond_3 & thinning_cond_4;

    uint8_t write_data = BINARY_WHITE + ((1 - thinning_cond_ok) * s_src[s_row * s_width + s_col]);
    global_mem_write(g_dst, g_row, g_col, g_width, g_height, write_data);
}

// Computes the number of white to black transitions around a pixel.
__device__ uint8_t wb_transitions_around(uint8_t* s_data, int s_row, int s_col, int s_width) {
    uint8_t count = 0;

    count += ((P2_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P3_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P3_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P4_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P4_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P5_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P5_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P6_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P6_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P7_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P7_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P8_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P8_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P9_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));
    count += ((P9_f(s_data, s_row, s_col, s_width) == BINARY_WHITE) & (P2_f(s_data, s_row, s_col, s_width) == BINARY_BLACK));

    return count;
}

int main(int argc, char** argv) {
    Bitmap* src_bitmap = NULL;
    Bitmap* dst_bitmap = NULL;
    dim3 grid_dim;
    dim3 block_dim;

    gpu_pre_skeletonization(argc, argv, &src_bitmap, &dst_bitmap, &grid_dim, &block_dim);

    int iterations = skeletonize(&src_bitmap, &dst_bitmap, grid_dim, block_dim);
    printf(" %u iterations\n", iterations);
    printf("\n");

    gpu_post_skeletonization(argv, &src_bitmap, &dst_bitmap);

    return EXIT_SUCCESS;
}
