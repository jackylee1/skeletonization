#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "gpu3.cuh"
#include "../common/gpu_only_utils.cuh"
#include "../common/lspbmp.hpp"
#include "../common/utils.hpp"

#define PAD_TOP 2
#define PAD_LEFT 2
#define PAD_BOTTOM 1
#define PAD_RIGHT 1

void and_reduction(uint8_t* d_data, int width, int height, dim3 grid_dim, dim3 block_dim) {
    int shared_mem_size = block_dim.x * block_dim.y * sizeof(uint8_t);

    // iterative reductions of d_data
    do {
        and_reduction<<<grid_dim, block_dim, shared_mem_size>>>(d_data, width, height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        width = grid_dim.x;
        height = grid_dim.y;
        grid_dim.x = ceil(grid_dim.x / ((double) block_dim.x));
        grid_dim.y = ceil(grid_dim.y / ((double) block_dim.y));
    } while ((width * height) != 1);
}

// Adapted for 2D arrays from Nvidia cuda SDK samples
__global__ void and_reduction(uint8_t* d_data, int width, int height) {
    // shared memory for tile
    extern __shared__ uint8_t s_data[];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Load equality values into shared memory tile. We use 1 as the default
    // value, as it is an AND reduction
    s_data[tid] = is_outside_image(row, col, width, height) ? 1 : global_mem_read(d_data, row, col, width, height);;
    __syncthreads();

    // do reduction in shared memory
    for (int s = ((blockDim.x * blockDim.y) / 2); s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] &= s_data[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        int write_data = s_data[0];
        global_mem_write(d_data, blockIdx.y, blockIdx.x, gridDim.x, gridDim.y, write_data);
    }
}

// Computes the number of black neighbors around a pixel.
__device__ uint8_t black_neighbors_around(uint8_t* d_data, int row, int col, int width) {
    uint8_t count = 0;

    count += (P2_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P3_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P4_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P5_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P6_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P7_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P8_f(d_data, row, col, width) == BINARY_BLACK);
    count += (P9_f(d_data, row, col, width) == BINARY_BLACK);

    return count;
}

__device__ uint8_t global_mem_read(uint8_t* d_data, int row, int col, int width, int height) {
    return is_outside_image(row, col, width, height) ? BINARY_WHITE : d_data[row * width + col];
}

__device__ void global_mem_write(uint8_t* d_data, int row, int col, int width, int height, uint8_t write_data) {
    if (!is_outside_image(row, col, width, height)) {
        d_data[row * width + col] = write_data;
    }
}

__device__ uint8_t is_outside_image(int row, int col, int width, int height) {
    return (row < 0) | (row > (height - 1)) | (col < 0) | (col > (width - 1));
}

__device__ uint8_t P2_f(uint8_t* data, int row, int col, int width) {
    return data[(row - 1) * width + col];
}

__device__ uint8_t P3_f(uint8_t* data, int row, int col, int width) {
    return data[(row - 1) * width + (col - 1)];
}

__device__ uint8_t P4_f(uint8_t* data, int row, int col, int width) {
    return data[row * width + (col - 1)];
}

__device__ uint8_t P5_f(uint8_t* data, int row, int col, int width) {
    return data[(row + 1) * width + (col - 1)];
}

__device__ uint8_t P6_f(uint8_t* data, int row, int col, int width) {
    return data[(row + 1) * width + col];
}

__device__ uint8_t P7_f(uint8_t* data, int row, int col, int width) {
    return data[(row + 1) * width + (col + 1)];
}

__device__ uint8_t P8_f(uint8_t* data, int row, int col, int width) {
    return data[row * width + (col + 1)];
}

__device__ uint8_t P9_f(uint8_t* data, int row, int col, int width) {
    return data[(row - 1) * width + (col + 1)];
}

__global__ void pixel_equality(uint8_t* d_in_1, uint8_t* d_in_2, uint8_t* d_out, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int write_data = (global_mem_read(d_in_1, row, col, width, height) == global_mem_read(d_in_2, row, col, width, height));
    global_mem_write(d_out, row, col, width, height, write_data);
}

// Performs an image skeletonization algorithm on the input Bitmap, and stores
// the result in the output Bitmap.
int skeletonize(Bitmap** src_bitmap, Bitmap** dst_bitmap, dim3 grid_dim, dim3 block_dim) {
    // allocate memory on device
    uint8_t* d_src_data = NULL;
    uint8_t* d_dst_data = NULL;
    uint8_t* d_equ_data = NULL;
    int data_size = (*src_bitmap)->width * (*src_bitmap)->height * sizeof(uint8_t);
    gpuErrchk(hipMalloc((void**) &d_src_data, data_size));
    gpuErrchk(hipMalloc((void**) &d_dst_data, data_size));
    gpuErrchk(hipMalloc((void**) &d_equ_data, data_size));

    // send data to device
    gpuErrchk(hipMemcpy(d_src_data, (*src_bitmap)->data, data_size, hipMemcpyHostToDevice));

    uint8_t are_identical_bitmaps = 0;
    int iterations = 0;
    do {
        int skeletonize_pass_shared_mem_size = (block_dim.x + PAD_LEFT + PAD_RIGHT) * (block_dim.y + PAD_TOP + PAD_BOTTOM) * sizeof(uint8_t);
        skeletonize_pass<<<grid_dim, block_dim, skeletonize_pass_shared_mem_size>>>(d_src_data, d_dst_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        pixel_equality<<<grid_dim, block_dim>>>(d_src_data, d_dst_data, d_equ_data, (*src_bitmap)->width, (*src_bitmap)->height);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        and_reduction(d_equ_data, (*src_bitmap)->width, (*src_bitmap)->height, grid_dim, block_dim);

        // bring reduced bitmap equality information back from device
        gpuErrchk(hipMemcpy(&are_identical_bitmaps, d_equ_data, 1 * sizeof(uint8_t), hipMemcpyDeviceToHost));

        swap_bitmaps((void**) &d_src_data, (void**) &d_dst_data);

        iterations++;
        printf(".");
        fflush(stdout);
    } while (!are_identical_bitmaps);

    // bring dst_bitmap back from device
    gpuErrchk(hipMemcpy((*dst_bitmap)->data, d_dst_data, data_size, hipMemcpyDeviceToHost));

    // free memory on device
    gpuErrchk(hipFree(d_src_data));
    gpuErrchk(hipFree(d_dst_data));
    gpuErrchk(hipFree(d_equ_data));

    return iterations;
}

// Performs 1 iteration of the thinning algorithm.
__global__ void skeletonize_pass(uint8_t* d_src, uint8_t* d_dst, int d_width, int d_height) {
    // shared memory for tile
    extern __shared__ uint8_t s_src[];

    int d_row = blockIdx.y * blockDim.y + threadIdx.y;
    int d_col = blockIdx.x * blockDim.x + threadIdx.x;

    int s_row = threadIdx.y + PAD_TOP;
    int s_col = threadIdx.x + PAD_LEFT;
    int s_width = blockDim.x + PAD_LEFT + PAD_RIGHT;

    // load data into shared memory
    if ((threadIdx.y == 0) & (threadIdx.x == 0)) {
        // top-left corner
        s_src[(s_row - 2) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row - 2, d_col - 2, d_width, d_height);
        s_src[(s_row - 2) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row - 2, d_col - 1, d_width, d_height);
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(d_src, d_row - 2, d_col, d_width, d_height);

        s_src[(s_row - 1) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row - 1, d_col - 2, d_width, d_height);
        s_src[(s_row - 1) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row - 1, d_col - 1, d_width, d_height);
        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(d_src, d_row - 1, d_col, d_width, d_height);

        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row, d_col - 2, d_width, d_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row, d_col - 1, d_width, d_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
    } else if ((threadIdx.y == (blockDim.y - 1)) & (threadIdx.x == 0)) {
        // bottom-left corner
        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row, d_col - 2, d_width, d_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row, d_col - 1, d_width, d_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);

        s_src[(s_row + 1) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row + 1, d_col - 2, d_width, d_height);
        s_src[(s_row + 1) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row + 1, d_col - 1, d_width, d_height);
        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(d_src, d_row + 1, d_col, d_width, d_height);
    } else if ((threadIdx.y == (blockDim.y - 1)) & (threadIdx.x == (blockDim.x - 1))) {
        // bottom-right corner
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row, d_col + 1, d_width, d_height);

        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(d_src, d_row + 1, d_col, d_width, d_height);
        s_src[(s_row + 1) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row + 1, d_col + 1, d_width, d_height);
    } else if ((threadIdx.y == 0) & (threadIdx.x == (blockDim.x - 1))) {
        // top-right corner
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(d_src, d_row - 2, d_col, d_width, d_height);
        s_src[(s_row - 2) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row - 2, d_col + 1, d_width, d_height);

        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(d_src, d_row - 1, d_col, d_width, d_height);
        s_src[(s_row - 1) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row - 1, d_col + 1, d_width, d_height);

        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row, d_col + 1, d_width, d_height);
    } else if (threadIdx.y == 0) {
        // PAD_TOP top rows
        s_src[(s_row - 2) * s_width + (s_col)] = global_mem_read(d_src, d_row - 2, d_col, d_width, d_height);
        s_src[(s_row - 1) * s_width + (s_col)] = global_mem_read(d_src, d_row - 1, d_col, d_width, d_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
    } else if (threadIdx.x == 0) {
        // PAD_LEFT left rows
        s_src[(s_row) * s_width + (s_col - 2)] = global_mem_read(d_src, d_row, d_col - 2, d_width, d_height);
        s_src[(s_row) * s_width + (s_col - 1)] = global_mem_read(d_src, d_row, d_col - 1, d_width, d_height);
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
    } else if (threadIdx.y == (blockDim.y - 1)) {
        // PAD_BOTTOM bottom rows
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
        s_src[(s_row + 1) * s_width + (s_col)] = global_mem_read(d_src, d_row + 1, d_col, d_width, d_height);
    } else if (threadIdx.x == (blockDim.x - 1)) {
        // PAD_RIGHT right rows
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
        s_src[(s_row) * s_width + (s_col + 1)] = global_mem_read(d_src, d_row, d_col + 1, d_width, d_height);
    } else {
        // center pixels
        s_src[(s_row) * s_width + (s_col)] = global_mem_read(d_src, d_row, d_col, d_width, d_height);
    }

    __syncthreads();

    uint8_t NZ = black_neighbors_around(s_src, s_row, s_col, s_width);
    uint8_t TR_P1 = wb_transitions_around(s_src, s_row, s_col, s_width);
    uint8_t TR_P2 = wb_transitions_around(s_src, s_row - 1, s_col, s_width);
    uint8_t TR_P4 = wb_transitions_around(s_src, s_row, s_col - 1, s_width);
    uint8_t P2 = P2_f(s_src, s_row, s_col, s_width);
    uint8_t P4 = P4_f(s_src, s_row, s_col, s_width);
    uint8_t P6 = P6_f(s_src, s_row, s_col, s_width);
    uint8_t P8 = P8_f(s_src, s_row, s_col, s_width);

    uint8_t thinning_cond_1 = ((2 <= NZ) & (NZ <= 6));
    uint8_t thinning_cond_2 = (TR_P1 == 1);
    uint8_t thinning_cond_3 = (((P2 & P4 & P8) == 0) | (TR_P2 != 1));
    uint8_t thinning_cond_4 = (((P2 & P4 & P6) == 0) | (TR_P4 != 1));
    uint8_t thinning_cond_ok = thinning_cond_1 & thinning_cond_2 & thinning_cond_3 & thinning_cond_4;

    uint8_t write_data = BINARY_WHITE + ((1 - thinning_cond_ok) * s_src[s_row * s_width + s_col]);
    global_mem_write(d_dst, d_row, d_col, d_width, d_height, write_data);
}

// Computes the number of white to black transitions around a pixel.
__device__ uint8_t wb_transitions_around(uint8_t* d_data, int row, int col, int width) {
    uint8_t count = 0;

    count += ((P2_f(d_data, row, col, width) == BINARY_WHITE) & (P3_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P3_f(d_data, row, col, width) == BINARY_WHITE) & (P4_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P4_f(d_data, row, col, width) == BINARY_WHITE) & (P5_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P5_f(d_data, row, col, width) == BINARY_WHITE) & (P6_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P6_f(d_data, row, col, width) == BINARY_WHITE) & (P7_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P7_f(d_data, row, col, width) == BINARY_WHITE) & (P8_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P8_f(d_data, row, col, width) == BINARY_WHITE) & (P9_f(d_data, row, col, width) == BINARY_BLACK));
    count += ((P9_f(d_data, row, col, width) == BINARY_WHITE) & (P2_f(d_data, row, col, width) == BINARY_BLACK));

    return count;
}

int main(int argc, char** argv) {
    Bitmap* src_bitmap = NULL;
    Bitmap* dst_bitmap = NULL;
    dim3 grid_dim;
    dim3 block_dim;

    gpu_pre_skeletonization(argc, argv, &src_bitmap, &dst_bitmap, &grid_dim, &block_dim);

    int iterations = skeletonize(&src_bitmap, &dst_bitmap, grid_dim, block_dim);
    printf(" %u iterations\n", iterations);
    printf("\n");

    gpu_post_skeletonization(argv, &src_bitmap, &dst_bitmap);

    return EXIT_SUCCESS;
}
